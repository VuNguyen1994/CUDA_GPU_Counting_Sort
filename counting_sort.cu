#include "hip/hip_runtime.h"
/* Host-side code to perform counting sort 
 * 
 * Author: Naga Kandasamy
 * Date modified: March 2, 2021
 * 
 * Student name(s): Dinh Nguyen, Tri Pham, Manh Cuong Phi
 * Date modified: 03/14/2021
 * 
 * Compile as follows: make clean && make
 */

#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <string.h>
#include <math.h>
#include <limits.h>

#include "counting_sort_kernel.cu"

/* Do not change the range value */
#define MIN_VALUE 0 
#define MAX_VALUE 255
#define THREAD_BLOCK_SIZE 256 
#define NUM_BLOCKS 4 
#define HISTOGRAM_SIZE 256 /* Histogram has 256 bins */

/* Uncomment to spit out debug info */
// #define DEBUG

extern "C" int counting_sort_gold(int *, int *, int, int);
int rand_int(int, int);
void print_array(int *, int);
void print_min_and_max_in_array(int *, int);
void compute_on_device(int *, int *, int, int);
int check_if_sorted(int *, int);
int compare_results(int *, int *, int);
void check_for_error(const char *);

int main(int argc, char **argv)
{
    if (argc < 2) {
        printf("Usage: %s num-elements\n", argv[0]);
        exit(EXIT_FAILURE);
    }

    int num_elements = atoi(argv[1]);
    int range = MAX_VALUE - MIN_VALUE;
    int *input_array, *sorted_array_reference, *sorted_array_d;

    /* Populate input array with random integers between [0, RANGE] */
    printf("Generating input array with %d elements in the range 0 to %d\n", num_elements, range);
    input_array = (int *)malloc(num_elements * sizeof(int));
    if (input_array == NULL) {
        perror("malloc");
        exit(EXIT_FAILURE);
    }
    
    srand(time(NULL));
    int i;
    for (i = 0; i < num_elements; i++)
        input_array[i] = rand_int (MIN_VALUE, MAX_VALUE);

#ifdef DEBUG
    print_array(input_array, num_elements);
    print_min_and_max_in_array(input_array, num_elements);
#endif

    struct timeval start, stop;

    /* Sort elements in input array using reference implementation. 
     * The result is placed in sorted_array_reference. */
    printf("\nSorting array on CPU\n");
    int status;
    sorted_array_reference = (int *)malloc(num_elements * sizeof(int));
    if (sorted_array_reference == NULL) {
        perror("malloc"); 
        exit(EXIT_FAILURE);
    }
    memset(sorted_array_reference, 0, num_elements);
    gettimeofday(&start, NULL);
    status = counting_sort_gold(input_array, sorted_array_reference, num_elements, range);
    gettimeofday(&stop, NULL);
    if (status == -1) {
        exit(EXIT_FAILURE);
    }

    status = check_if_sorted(sorted_array_reference, num_elements);
    if (status == -1) {
        printf("Error sorting the input array using the reference code\n");
        exit(EXIT_FAILURE);
    }

    printf("Counting sort was successful on the CPU\n");
    fprintf(stderr, "CPU Execution time = %fs\n", (float)(stop.tv_sec - start.tv_sec +(stop.tv_usec - start.tv_usec)/(float)1000000));

#ifdef DEBUG
    print_array(sorted_array_reference, num_elements);
#endif

    /* FIXME: Write function to sort elements in the array in parallel fashion. 
     * The result should be placed in sorted_array_mt. */
    printf("\nSorting array on GPU\n");
    sorted_array_d = (int *)malloc(num_elements * sizeof(int));
    if (sorted_array_d == NULL) {
        perror("malloc");
        exit(EXIT_FAILURE);
    }
    memset(sorted_array_d, 0, num_elements);
    compute_on_device(input_array, sorted_array_d, num_elements, range);

#ifdef DEBUG
    print_array(sorted_array_d, num_elements);
#endif
    /* Check the two results for correctness */
    printf("\nComparing CPU and GPU results\n");
    status = compare_results(sorted_array_reference, sorted_array_d, num_elements);
    if (status == 0)
        printf("Test passed\n");
    else
        printf("Test failed\n");

    exit(EXIT_SUCCESS);
}


/* FIXME: Write the GPU implementation of counting sort */
void compute_on_device(int *input_array, int *sorted_array, int num_elements, int range)
{
    struct timeval start, stop;

    int *input_array_on_device = NULL;
	int *sorted_array_on_device = NULL;
    int *prefix_array = (int *)malloc(HISTOGRAM_SIZE * sizeof(int));
    int *prefix_array_on_device = NULL;

    /* Set up the execution grid on GPU */
	dim3 thread_block(THREAD_BLOCK_SIZE, 1);
	dim3 grid(NUM_BLOCKS,1);

    
    /* Allocate space on GPU for input data */
	hipMalloc((void**)&input_array_on_device, num_elements * sizeof(int));
	hipMemcpy(input_array_on_device, input_array, num_elements * sizeof(int), hipMemcpyHostToDevice);

    /* Allocate space on GPU  initialize contents to zero */
	hipMalloc((void**)&sorted_array_on_device, num_elements * sizeof(int));
	hipMemset(sorted_array_on_device, 0, num_elements * sizeof(int));

	hipMalloc((void**)&prefix_array_on_device, HISTOGRAM_SIZE * sizeof(int));
	hipMemset(prefix_array_on_device, 0, HISTOGRAM_SIZE * sizeof(int));

    gettimeofday(&start, NULL);
    // Launch kernel to find prefix array
    find_prefix_kernel<<<grid, thread_block>>>(input_array_on_device, prefix_array_on_device, num_elements, range);
    hipDeviceSynchronize();
    // Launch kernel to form sorted array using the prefix array as input
    counting_sort_kernel<<<grid,thread_block>>>(prefix_array_on_device, sorted_array_on_device, num_elements, range);
    hipDeviceSynchronize();

    gettimeofday(&stop, NULL);
	fprintf(stderr, "GPU Execution time = %fs\n", (float)(stop.tv_sec - start.tv_sec +(stop.tv_usec - start.tv_usec)/(float)1000000));

    /* Copy result back from GPU */ 
	hipMemcpy(sorted_array, sorted_array_on_device, num_elements * sizeof(int), hipMemcpyDeviceToHost);

    check_for_error("KERNEL FAILURE");

    /* Free memory */
	hipFree(input_array_on_device);
	hipFree(sorted_array_on_device);
    hipFree(prefix_array_on_device);
    free(prefix_array);

    return;
}

/* Check for errors during kernel execution */
void check_for_error(const char *msg)
{
	hipError_t err = hipGetLastError();
	if (hipSuccess != err) {
		printf("CUDA ERROR: %s (%s)\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
} 

/* Check if array is sorted */
int check_if_sorted(int *array, int num_elements)
{
    int status = 0;
    int i;
    for (i = 1; i < num_elements; i++) {
        if (array[i - 1] > array[i]) {
            status = -1;
            break;
        }
    }

    return status;
}

/* Check if the arrays elements are identical */ 
int compare_results(int *array_1, int *array_2, int num_elements)
{
    int status = 0;
    int i;
    for (i = 0; i < num_elements; i++) {
        if (array_1[i] != array_2[i]) {
            status = -1;
            break;
        }
    }

    return status;
}

/* Return random integer between [min, max] */ 
int rand_int(int min, int max)
{
    float r = rand()/(float)RAND_MAX;
    return (int)floorf(min + (max - min) * r);
}

/* Print given array */
void print_array(int *this_array, int num_elements)
{
    printf("Array: ");
    int i;
    for (i = 0; i < num_elements; i++)
        printf("%d ", this_array[i]);
    
    printf("\n");
    return;
}

/* Return min and max values in given array */
void print_min_and_max_in_array(int *this_array, int num_elements)
{
    int i;

    int current_min = INT_MAX;
    for (i = 0; i < num_elements; i++)
        if (this_array[i] < current_min)
            current_min = this_array[i];

    int current_max = INT_MIN;
    for (i = 0; i < num_elements; i++)
        if (this_array[i] > current_max)
            current_max = this_array[i];

    printf("Minimum value in the array = %d\n", current_min);
    printf("Maximum value in the array = %d\n", current_max);
    return;
}


